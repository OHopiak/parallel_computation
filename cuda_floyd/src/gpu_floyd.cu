#include "hip/hip_runtime.h"
//
// Created by orest on 26/10/18.
//

#include "gpu_floyd.h"

static __global__
void gpu_floyd_kernel(const int u, size_t pitch, const int nvertex, int* const graph, int* const pred)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (y < nvertex && x < nvertex) {
		size_t indexYX = y * pitch + x;
		size_t indexUX = u * pitch + x;

		int newPath = graph[y * pitch + u] + graph[indexUX];
		int oldPath = graph[indexYX];
		if (oldPath > newPath) {
			graph[indexYX] = newPath;
			pred[indexYX] = pred[indexUX];
		}
	}
}

void gpu_floyd(const std::unique_ptr<graphFloydTopology>& dataHost)
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	HANDLE_ERROR(hipSetDevice(0));
	unsigned int nvertex = dataHost->nvertex;

	// Initialize the grid and block dimensions here
	dim3 dimGrid((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1, 1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

	int* graphDevice, * predDevice;
	size_t pitch = cudaMoveMemoryToDevice(dataHost, &graphDevice, &predDevice);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_floyd_kernel), hipFuncCachePreferL1);
	for (int vertex = 0; vertex < nvertex; ++vertex) {
		gpu_floyd_kernel<<<dimGrid, dimBlock>>>(vertex, pitch / sizeof(size_t), nvertex, graphDevice, predDevice);
	}

	// Check for any errors launching the kernel
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());
	cudaMoveMemoryToHost(graphDevice, predDevice, dataHost, pitch);
}

size_t cudaMoveMemoryToDevice(const std::unique_ptr<graphFloydTopology>& dataHost, int** graphDevice, int** predDevice)
{
	size_t height = dataHost->nvertex;
	size_t width = height * sizeof(int);
	size_t pitch;

	// Allocate GPU buffers for matrix of shortest paths d(G) and predecessors p(G)
	HANDLE_ERROR(hipMallocPitch(graphDevice, &pitch, width, height));
	HANDLE_ERROR(hipMallocPitch(predDevice, &pitch, width, height));

	// Copy input from host memory to GPU buffers and
	HANDLE_ERROR(hipMemcpy2D(*graphDevice, pitch,
							  dataHost->graph.get(), width, width, height, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy2D(*predDevice, pitch,
							  dataHost->pred.get(), width, width, height, hipMemcpyHostToDevice));

	return pitch;
}

void cudaMoveMemoryToHost(int* graphDevice, int* predDevice, const std::unique_ptr<graphFloydTopology>& dataHost,
						  size_t pitch)
{
	size_t height = dataHost->nvertex;
	size_t width = height * sizeof(int);

	HANDLE_ERROR(hipMemcpy2D(dataHost->pred.get(), width, predDevice, pitch, width, height, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy2D(dataHost->graph.get(), width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(predDevice));
	HANDLE_ERROR(hipFree(graphDevice));
}